#include "hip/hip_runtime.h"
// **************************************************************************
//                                   lj.cu
//                             -------------------
//                           W. Michael Brown (ORNL)
//
//  Device code for acceleration of the lj/cut pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : brownw@ornl.gov
// ***************************************************************************

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "lal_aux_fun1.h"

#if defined(NV_KERNEL) || defined(USE_HIP)
_texture( pos_tex,float4);
#else
#define pos_tex x_
#endif

#define fast_mul(X,Y) (X)*(Y)

#define fetch4(lans,i,pos_tex) {			\
    int4 xy = tex1Dfetch(pos_tex,i*2);		\
    int4 zt = tex1Dfetch(pos_tex,i*2+1);	\
    ans.x=__hiloint2double(xy.y, xy.x);		\
    ans.y=__hiloint2double(xy.w, xy.z);		\
    ans.z=__hiloint2double(zt.y, zt.x);		\
    ans.w=__hiloint2double(zt.w, zt.z);		\
  }
#define fetch(ans,i,q_tex) {			\
    int2 qt = tex1Dfetch(q_tex,i);		\
    ans=__hiloint2double(qt.y, qt.x);		\
  }

#define __kernel extern "C" __global__
#define __local __shared__
#define __global
#define atom_add atomicAdd
#define ucl_inline static __inline__ __device__

#define numtyp double
#define numtyp2 double2
#define numtyp4 double4
#define acctyp double
#define acctyp2 double2
#define acctyp4 double4

#define SBBITS 30
#define NEIGHMASK 0x3FFFFFFF
ucl_inline int sbmask(int j) { return j >> SBBITS & 3; };

#define MAX_SHARED_TYPES 8
#define restrict __restrict__
#define BLOCK_PAIR 256
#define SIMD_SIZE 32
#define simd_size() SIMD_SIZE

#define EVFLAG 1
#define NOUNROLL
#define GLOBAL_ID_X threadIdx.x+fast_mul(blockIdx.x,blockDim.x)
#define GLOBAL_ID_Y threadIdx.y+fast_mul(blockIdx.y,blockDim.y)
#define GLOBAL_SIZE_X fast_mul(gridDim.x,blockDim.x);
#define GLOBAL_SIZE_Y fast_mul(gridDim.y,blockDim.y);
#define THREAD_ID_X threadIdx.x
#define THREAD_ID_Y threadIdx.y
#define BLOCK_ID_X blockIdx.x
#define BLOCK_ID_Y blockIdx.y
#define BLOCK_SIZE_X blockDim.x
#define BLOCK_SIZE_Y blockDim.y
#define NUM_BLOCKS_X gridDim.x

#define simdsync() __syncwarp(0xffffffff)

__kernel void k_lj_fast(const __global numtyp4* restrict x_,
                        const __global numtyp4* restrict lj1_in,
                        const __global numtyp4* restrict lj3_in,
                        const __global numtyp* restrict sp_lj_in,
                        const __global int* dev_nbor,
                        const __global int* dev_packed,
                        __global acctyp4* restrict ans,
                        __global acctyp* restrict engv,
                        const int eflag, const int vflag, const int inum,
                        const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

#ifndef ONETYPE
  __local numtyp4 lj1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 lj3[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[4];
  if (tid<4)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    lj1[tid]=lj1_in[tid];
    if (EVFLAG && eflag)
      lj3[tid]=lj3_in[tid];
  }
  __syncthreads();
  #else
  const numtyp lj1x=lj1_in[ONETYPE].x;
  const numtyp lj1y=lj1_in[ONETYPE].y;
  const numtyp cutsq=lj1_in[ONETYPE].z;
  numtyp lj3x, lj3y, lj3z;
  if (EVFLAG && eflag) {
    lj3x=lj3_in[ONETYPE].x;
    lj3y=lj3_in[ONETYPE].y;
    lj3z=lj3_in[ONETYPE].z;
  }
  #endif

  int n_stride;
  local_allocate_store_pair();

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    #ifndef ONETYPE
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);
    numtyp factor_lj;
    #endif

    NOUNROLL
    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];
      #ifndef ONETYPE
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;
      #endif

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      #ifndef ONETYPE
      int mtype=itype+jx.w;
      numtyp cutsq=lj1[mtype].z;
      #endif

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp r2inv = delx*delx+dely*dely+delz*delz;

      if (r2inv<cutsq) {
        #ifndef ONETYPE
        numtyp lj1x=lj1[mtype].x;
        numtyp lj1y=lj1[mtype].y;
        #endif

        r2inv=ucl_recip(r2inv);
        numtyp r6inv = r2inv*r2inv*r2inv;
        numtyp force = r2inv*r6inv*(lj1x*r6inv-lj1y);
        #ifndef ONETYPE
        force*=factor_lj;
        #endif

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag) {
          #ifndef ONETYPE
          numtyp lj3x=lj3[mtype].x;
          numtyp lj3y=lj3[mtype].y;
          numtyp lj3z=lj3[mtype].z;
          #endif
          numtyp e=r6inv*(lj3x*r6inv-lj3y);
          #ifndef ONETYPE
          energy+=factor_lj*(e-lj3z);
          #else
          energy+=(e-lj3z);
          #endif
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
}

