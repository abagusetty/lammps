#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 32
#define M 128

// texture object is a kernel argument
__global__ void kernel(hipTextureObject_t tex) {
  int i = blockIdx.x *blockDim.x + threadIdx.x;
  float x = tex1Dfetch<float>(tex, i);
  if (i < 256) printf("%d %f\n", i, x);
}

void call_kernel(hipTextureObject_t tex) {
  dim3 block(128,1,1);
  dim3 grid((N*M)/block.x,1,1);
  kernel <<<grid, block>>>(tex);
}

int main() {
  // declare and allocate memory
  float *buffer, *h_buffer;
  size_t pitch;
  hipMallocPitch(&buffer, &pitch, N*sizeof(float),M);
  printf("pitch = %lu\n", pitch);
  hipMemset(buffer, 0, M*pitch);
  h_buffer=(float *)malloc(N*M*sizeof(float));
  for (int i = 0; i < M; i++)
    for (int j = 0; j < N; j++) h_buffer[i*N+j] = i+1;
  hipMemcpy2D(buffer, pitch, h_buffer, N*sizeof(float), N*sizeof(float), M,  hipMemcpyHostToDevice);
  // create texture object
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = buffer;
  resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
  resDesc.res.linear.desc.x = 32; // bits per channel
  resDesc.res.linear.sizeInBytes = M*pitch*sizeof(float);

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;

  // create texture object: we only have to do this once!
  hipTextureObject_t tex=0;
  hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

  call_kernel(tex); // pass texture as argument

  // destroy texture object
  hipDestroyTextureObject(tex);

  hipFree(buffer);
}